#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void add(int *a, int *b, int *c) {
    c[0] = a[0] + b[0];
}

int main(void) {
    int a, b, c;
    int *d_a, *d_b, *d_c;
    int size = sizeof(int);
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);
    a = 3;
    b = 5;

    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    add<<<1, 1>>>(d_a, d_b, d_c);
    hipDeviceSynchronize();
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
    printf("Result: %d\n", c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
