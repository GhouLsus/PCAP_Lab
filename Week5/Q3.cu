#include <iostream>
#include <hip/hip_runtime.h>
#include <cmath>

__global__ void computeSine(float *angles, float *sine_values, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        sine_values[idx] = sinf(angles[idx]);
    }
}

int main() {
    int N = 10;
    int size = N * sizeof(float);  

    float *h_angles = new float[N];
    float *h_sine_values = new float[N];
    for (int i = 0; i < N; i++) {
        h_angles[i] = i * M_PI / 5;
    }
    float *d_angles, *d_sine_values;
    hipMalloc((void**)&d_angles, size);
    hipMalloc((void**)&d_sine_values, size);
    hipMemcpy(d_angles, h_angles, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;

    int numBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    computeSine<<<numBlocks, threadsPerBlock>>>(d_angles, d_sine_values, N);
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "CUDA kernel launch failed: " << hipGetErrorString(error) << std::endl;
        return -1;
    }
    hipMemcpy(h_sine_values, d_sine_values, size, hipMemcpyDeviceToHost);
    std::cout << "Angles in radians and their sine values:\n";
    for (int i = 0; i < N; i++) {
        std::cout << "Angle: " << h_angles[i] << "  Sine: " << h_sine_values[i] << std::endl;
    }
    hipFree(d_angles);
    hipFree(d_sine_values);
    delete[] h_angles;
    delete[] h_sine_values;

    return 0;
}
