#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define N 1024

__global__ void CUDACount(char* A, unsigned int *d_count, int str_length) {
    int i = threadIdx.x;
    if (i < str_length) { 
        if (A[i] == 'a')  
            atomicAdd(d_count, 1);
    }
}

int main() {
    char A[N];
    char *d_A;
    unsigned int count = 0, *d_count, result = 0;  
    printf("Enter a string: ");
    fgets(A, N, stdin);  


    A[strcspn(A, "\n")] = 0;

    int str_length = strlen(A);

    if (str_length >= N) {
        printf("Error: Input string is too long!\n");
        return -1;
    }

    if (hipMalloc((void**)&d_A, (str_length + 1) * sizeof(char)) != hipSuccess ||
        hipMalloc((void**)&d_count, sizeof(unsigned int)) != hipSuccess) {
        printf("CUDA Memory Allocation Error\n");
        return -1;
    }

    hipMemcpy(d_A, A, (str_length + 1) * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_count, &count, sizeof(unsigned int), hipMemcpyHostToDevice);

    CUDACount<<<1, str_length>>>(d_A, d_count, str_length);
    hipDeviceSynchronize();  
    hipMemcpy(&result, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost);
    printf("Total occurrences of 'a': %u\n", result);
    hipFree(d_A);
    hipFree(d_count);

    return 0;
}
